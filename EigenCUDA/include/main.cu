#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"



__global__ void exec_kernel()
{

}

TEST(Test, exe)
{
    exec_kernel<<<1, 1>>>();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);    
}


int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
