#include "hip/hip_runtime.h"
#include <Eigen/Dense>

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"



__device__ __host__ void test_func()
{
    Eigen::Matrix3d matrix;
    matrix << 1, 2, 3,  //
        4, 5, 6,        //
        7, 8, 9;

    Eigen::SelfAdjointEigenSolver<Eigen::Matrix3d> eigen_solver(matrix);

    Eigen::Vector3d eigenvalues  = eigen_solver.eigenvalues();
    Eigen::Matrix3d eigenvectors = eigen_solver.eigenvectors();


    printf("\n Original = {%f, %f, %f || %f, %f, %f || %f, %f, %f}",
           matrix(0, 0),
           matrix(0, 1),
           matrix(0, 2),
           matrix(1, 0),
           matrix(1, 1),
           matrix(1, 2),
           matrix(2, 0),
           matrix(2, 1),
           matrix(2, 2));


    printf("\n eigenvalues = {%f, %f, %f}",
           eigenvalues(0),
           eigenvalues(1),
           eigenvalues(2));

    // verify
    Eigen::Matrix3d reconstructed_matrix =
        eigenvectors * eigenvalues.asDiagonal() * eigenvectors.transpose();

    printf("\n Reconstr = {%f, %f, %f || %f, %f, %f || %f, %f, %f}",
           reconstructed_matrix(0, 0),
           reconstructed_matrix(0, 1),
           reconstructed_matrix(0, 2),
           reconstructed_matrix(1, 0),
           reconstructed_matrix(1, 1),
           reconstructed_matrix(1, 2),
           reconstructed_matrix(2, 0),
           reconstructed_matrix(2, 1),
           reconstructed_matrix(2, 2));

    printf("\n");
}

__global__ void eigen_decomp()
{
    printf("\n From the device");
    test_func();
}

TEST(Test, exe)
{
    eigen_decomp<<<1, 1>>>();
    auto err = hipDeviceSynchronize();
    EXPECT_EQ(err, hipSuccess);

    printf("\n From the host");
    test_func();
}


int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
